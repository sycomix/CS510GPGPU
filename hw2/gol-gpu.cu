#include "hip/hip_runtime.h"
/* Henry Cooney - CS510, Accel. Comp. - 4 July 2015

   Conway's game of life, computed on the GPU.
  
   Uses a tiled convolution pattern to achieve good performance
   (hopefully)
*/


#include "gol.h"

int main() {
  
  test_gol(1, HEIGHT, WIDTH);
  //  fill_board
  //gpu_compute(current, HEIGHT, WIDTH, 1);
  return 0;
}


int* gpu_compute(int* initial, int height, int width, int timesteps) {
  // Does GoL on the GPU. Initial is the starting
  // matrix (it is not modified.) The resulting matrix after timesteps
  // iterations is returned.

  printf("Launching GPU computation for %d timesteps... \n", timesteps);
  int n = width * height;
  int* result = (int*) malloc(sizeof(int) * n);
  int* current_dev,* next_dev;


  //  int tester[n];
  //int i;
  //for(i=0; i < n; ++i)
  // tester[i] = 0;
  
  // Memory transfer
  printCudaError(hipMalloc((void**) &current_dev, sizeof(int)*n));
  printCudaError(hipMalloc((void**) &next_dev, sizeof(int)*n));
  
  hipDeviceSynchronize(); // is this necessary? 
  printCudaError(hipMemcpy(current_dev, initial, sizeof(int)*n, hipMemcpyHostToDevice));
  
  // Establish dimms - these are for GTX 645

  dim3 dimBlock(TW, TW, 1);
  dim3 dimGrid(divideRoundUp(width, ETW), divideRoundUp(height, ETW), 1);

  printf("Matrix size (width x height): %d x %d\n", width, height);
  printf("Block dims (x, y, z): %d x %d x %d\n", dimBlock.x, dimBlock.y, dimBlock.z);
  printf("Grid dims (x, y, z): %d x %d x %d\n", dimGrid.x, dimGrid.y, dimGrid.z);
  
  printf("Starting kernel... \n");
  // For testing - zero GPU memory
  zeroMemory<<<dim3(divideRoundUp(n,512),1,1), dim3(512, 1, 1)>>>(next_dev);
  conway_step_kernel<<<dimGrid, dimBlock>>>(current_dev, next_dev, height, width);

  printf("Kernel done. \n");
  hipDeviceSynchronize();
  // Check for errors from the kernel:
  if(hipGetLastError() != hipSuccess) {
    printf("*** ERROR IN KERNEL *** \n");
    exit(1);
  }
  
  // Copy memory back and free GPU mem
  printCudaError(hipMemcpy(result, next_dev, sizeof(int)*n, hipMemcpyDeviceToHost));

  printCudaError(hipFree(current_dev));
  printCudaError(hipFree(next_dev));
  
  return result;
}


__global__ 
void conway_step_kernel(int* current_dev, int* next_dev, 
			int height, int width) {
  // Advances the game of life one timestep.
  // current_dev is the initial matrix, it is not modified. next_dev 
  // the next timestep (the result)
  
  __shared__ int dsm[TW][TW]; // Device Shared Memory
  
  // Each thread is responsbile for a. fetching one item
  // from global memory and b. writing one item to output matrix.

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i, ii;
  int num_neighbors;
  int next = 0;
  int this_pixel;
  

  // Each output pixel requires knowledge of neighboring pixels.
  // Thus, each tile has 'egde pixels' which are loaded into shared mem
  // but not written to. Values must be shifted by one to compensate for this
  // conditional expressions handle wraparound
  // Mod arithmetic implements wraparound
  
  int row = (by*ETW + ty + height - 1) % height;
  int col = (bx*ETW + tx + width - 1) % width;
  
  int trueRow = by*ETW + ty;
  int trueCol = bx*ETW + tx;
  
  this_pixel = current_dev[row*width + col];
  dsm[ty][tx] = this_pixel;

  __syncthreads();

  //if (row >= 0 && row < height && col >= 0 && col < width 
  if(tx > 0 && tx <= ETW && ty > 0 && ty <= ETW) {
    // This pixel is not an edge pixel, so figure out its value
    // in the next frame, and write it.
    
    // num_neighbors is the sum of all the neighboring cells. Since
    // the loop will pass through this pixel, I negate this pixels value.
    // Thus, this pixel does not contribute to the overall sum.
    num_neighbors = -this_pixel; 
    for(i=-1; i<2; ++i) {
	for(ii=-1; ii<2; ++ii) {
	  num_neighbors += dsm[ty+i][tx+ii];
	}
    }

    next = 0;
    if(num_neighbors == 3 || (num_neighbors == 2 && this_pixel))
      next = 1;
	
    next_dev[row*width + col] = next;
  }  
}

void printCudaError(hipError_t err) {
  // Checks the value of input error. If it does not
  // indicate success, prints an error message.
  
  if(err != hipSuccess) {
    printf("**** CUDA ERROR: ****\n");
    printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  
}


int divideRoundUp(int a, int b) {
  // Divides a by b, but rounds the result up instead of down.
  return (a+(b-1)) / b;
}


__global__
void zeroMemory(int* ptr) {
  // Zeros out memory at a pointer.
  // For testing, since garbage hanging out in the GPU
  // can cause confusing results
  
  ptr[blockIdx.x*blockDim.x + threadIdx.x] = 0;
 
}
